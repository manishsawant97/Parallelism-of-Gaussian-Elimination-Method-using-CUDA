#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#include<math.h>

__global__ void Gaussian(float *input_d, float *output_d, int u_size){

	int idx = threadIdx.x;
	int idy = threadIdx.y;

	__shared__ float S[16][16];

	S[idy][idx] = input_d[(idy * (u_size + 1)) + idx];

	for (int i = 1; i < u_size; i++)
	{
		if ((idy + i) < u_size)
		{
			float Q = (-1)*(S[i - 1][i - 1] / S[i + idy][i - 1]);
			S[i + idy][idx] = S[i - 1][idx] + ((Q)* (S[i + idy][idx]));
		}
		__syncthreads();
	}

	output_d[idy * (u_size + 1) + idx] = S[idy][idx];
}

int main(int argc, char **argv){
	float *output_h = NULL;
	float *input_d,
	float *output_d;
	float *op_res, add, r;
	int unknowns, j;

	unknowns = 10;

	float input_h[11][10] = { 5393, 1190, 7142, 5286, 3511, 6181, 8170, 4859, 4792, 2336, 2362, 7596, 9238, 1654, 7061, 1840, 6848, 1481, 4925, 7249, 4966, 4472, 3297, 6822, 6247, 0627, 7489, 9559, 2693, 9945, 2189, 5459, 5647, 9772, 0052, 1868, 6421, 2763, 8424, 1441, 7702, 8918, 2616, 4178, 2148, 8236, 7316, 7422, 7102, 8063, 5179, 3831, 5220, 1054, 6207, 3404, 6501, 7209, 7743, 6141, 1947, 6527, 2443, 9385, 8557, 6354, 0306, 8450, 6675, 9345, 1334, 8023, 7956, 5500, 2870, 8104, 2402, 4818, 7570, 4004, 7356, 4645, 5433, 0413, 8130, 7636, 6461, 9986, 2148, 8236, -7316, 7422, 7102, 8063, 5179, 3831, -5220, 1054, -6207, 5032, 6929, 5927, 1588, 4605, 8211, 2082, 5780, 4686, 2656, 8745 };

	output_h = (float*)malloc(sizeof(float)*unknowns*(unknowns + 1));

	hipMalloc(&input_d, sizeof(float)*(unknowns)*(unknowns + 1));
	hipMalloc(&output_d, sizeof(float)*(unknowns)*(unknowns + 1));

	hipMemcpy(input_d, input_h, sizeof(float)*unknowns*(unknowns + 1), hipMemcpyHostToDevice);

	dim3 dimBlock(unknowns + 1, unknowns, 1);
	dim3 dimGrid(1, 1, 1);

	hipEvent_t startEvent, stopEvent; 
	hipEventCreate(&startEvent); 
	hipEventCreate(&stopEvent); 
	float ms;

	hipEventRecord(startEvent, 0);

	Gaussian << < dimGrid, dimBlock >> >(input_d, output_d, unknowns);

	hipEventRecord(stopEvent, 0);

	hipEventSynchronize(stopEvent);
	hipEventElapsedTime(&ms, startEvent, stopEvent);

	hipMemcpy(output_h, output_d, sizeof(float)*unknowns*(unknowns + 1), hipMemcpyDeviceToHost);
	

	//data coming from GPU 
	printf("\nOutput from GPU\n\n");

	for (int i = 0; i< unknowns; i++)
	{
		for (int j = 0; j< unknowns + 1; j++)
		{
			printf("%f\n", output_h[i*(unknowns + 1) + j]);
		}
		printf("\n");
	}

	//Back substitution 
	op_res = (float*)malloc(sizeof(float)*(unknowns));
	for (int i = 0; i< unknowns; i++)
	{
		op_res[i] = 1.0;
	}

	for (int i = unknowns - 1; i >= 0; i--)
	{
		add = 0.0;

		for (j = unknowns - 1; j>i; j--)
		{
			add = add + op_res[j] * output_h[i*(unknowns + 1) + j];
		}
		r = output_h[i*(unknowns + 1) + unknowns] - add;
		op_res[i] = r / output_h[i *(unknowns + 1) + j];
	}

	//Displaying the Unknown Variables
	printf("\n\t\tUNKNOWNS\n\n");
	for (int i = 0; i<unknowns; i++)
	{
		printf("[x%d] = %+f\n", i, op_res[i]);
	}

	//Print Execution Time
	printf("\nKernel Performance...\n\n"); 
	printf("Execution Time = %f miliseconds \n\n", ms);

	free(input_h);
	free(output_h);
	hipFree(input_d);
	hipFree(output_d);
	hipDeviceReset();
	return 0;
}